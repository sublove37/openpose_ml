#include "hip/hip_runtime.h"
#include <openpose/utilities/cuda.hpp>
#include <openpose/utilities/cuda.hu>
#include <openpose/core/resizeAndMergeBase.hpp>

namespace op
{
    const auto THREADS_PER_BLOCK_1D = 16u;

    template <typename T>
    __global__ void resizeKernel(T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight,
                                 const int targetWidth, const int targetHeight)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + 0.5f) * sourceWidth / T(targetWidth) - 0.5f;
            const T ySource = (y + 0.5f) * sourceHeight / T(targetHeight) - 0.5f;
            targetPtr[y*targetWidth+x] = bicubicInterpolate(sourcePtr, xSource, ySource, sourceWidth, sourceHeight,
                                                            sourceWidth);
        }
    }

    template <typename T>
    __global__ void resizeKernelAndMerge(T* targetPtr, const T* const sourcePtr, const T scaleWidth,
                                         const T scaleHeight, const int sourceWidth, const int sourceHeight,
                                         const int targetWidth, const int targetHeight, const int averageCounter)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + 0.5f) / scaleWidth - 0.5f;
            const T ySource = (y + 0.5f) / scaleHeight - 0.5f;
            const auto interpolated = bicubicInterpolate(sourcePtr, xSource, ySource, sourceWidth, sourceHeight,
                                                         sourceWidth);
            auto& targetPixel = targetPtr[y*targetWidth+x];
            targetPixel = ((averageCounter * targetPixel) + interpolated) / T(averageCounter + 1);
            // targetPixel = fastMax(targetPixel, interpolated);
        }
    }

    template <typename T>
    void resizeAndMergeGpu(T* targetPtr, const std::vector<const T*>& sourcePtrs, const std::array<int, 4>& targetSize,
                           const std::vector<std::array<int, 4>>& sourceSizes,
                           const std::vector<T>& scaleInputToNetInputs)
    {
        try
        {
            // Security checks
            if (sourceSizes.empty())
                error("sourceSizes cannot be empty.", __LINE__, __FUNCTION__, __FILE__);
            if (sourcePtrs.size() != sourceSizes.size() || sourceSizes.size() != scaleInputToNetInputs.size())
                error("Size(sourcePtrs) must match size(sourceSizes) and size(scaleInputToNetInputs). Currently: "
                      + std::to_string(sourcePtrs.size()) + " vs. " + std::to_string(sourceSizes.size()) + " vs. "
                      + std::to_string(scaleInputToNetInputs.size()) + ".", __LINE__, __FUNCTION__, __FILE__);

            // Parameters
            const auto channels = targetSize[1];
            const auto targetHeight = targetSize[2];
            const auto targetWidth = targetSize[3];
            const dim3 threadsPerBlock{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D};
            const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
                                 getNumberCudaBlocks(targetHeight, threadsPerBlock.y)};
            const auto& sourceSize = sourceSizes[0];
            const auto sourceHeight = sourceSize[2];
            const auto sourceWidth = sourceSize[3];

            // No multi-scale merging or no merging required
            if (sourceSizes.size() == 1)
            {
                const auto num = sourceSize[0];
                if (targetSize[0] > 1 || num == 1)
                {
                    const auto sourceChannelOffset = sourceHeight * sourceWidth;
                    const auto targetChannelOffset = targetWidth * targetHeight;
                    for (auto n = 0; n < num; n++)
                    {
                        const auto offsetBase = n*channels;
                        for (auto c = 0 ; c < channels ; c++)
                        {
                            const auto offset = offsetBase + c;
                            resizeKernel<<<numBlocks, threadsPerBlock>>>(targetPtr + offset * targetChannelOffset,
                                                                         sourcePtrs.at(0) + offset * sourceChannelOffset,
                                                                         sourceWidth, sourceHeight, targetWidth,
                                                                         targetHeight);
                        }
                    }
                }
                // Old inefficient multi-scale merging
                else
                    error("It should never reaches this point. Notify us.", __LINE__, __FUNCTION__, __FILE__);
            }
            // Multi-scaling merging
            else
            {
                const auto targetChannelOffset = targetWidth * targetHeight;
                hipMemset(targetPtr, 0.f, channels*targetChannelOffset * sizeof(T));
                auto averageCounter = -1;
                const auto scaleToMainScaleWidth = targetWidth / T(sourceWidth);
                const auto scaleToMainScaleHeight = targetHeight / T(sourceHeight);

                for (auto i = 0u ; i < sourceSizes.size(); i++)
                {
                    const auto& currentSize = sourceSizes.at(i);
                    const auto currentHeight = currentSize[2];
                    const auto currentWidth = currentSize[3];
                    const auto sourceChannelOffset = currentHeight * currentWidth;
                    const auto scaleInputToNet = scaleInputToNetInputs[i] / scaleInputToNetInputs[0];
                    const auto scaleWidth = scaleToMainScaleWidth / scaleInputToNet;
                    const auto scaleHeight = scaleToMainScaleHeight / scaleInputToNet;
                    averageCounter++;
                    for (auto c = 0 ; c < channels ; c++)
                    {
                        resizeKernelAndMerge<<<numBlocks, threadsPerBlock>>>(
                            targetPtr + c * targetChannelOffset, sourcePtrs[i] + c * sourceChannelOffset,
                            scaleWidth, scaleHeight, currentWidth, currentHeight, targetWidth,
                            targetHeight, averageCounter
                        );
                    }
                }
            }

            cudaCheck(__LINE__, __FUNCTION__, __FILE__);
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }

    template void resizeAndMergeGpu(float* targetPtr, const std::vector<const float*>& sourcePtrs,
                                    const std::array<int, 4>& targetSize,
                                    const std::vector<std::array<int, 4>>& sourceSizes,
                                    const std::vector<float>& scaleInputToNetInputs);
    template void resizeAndMergeGpu(double* targetPtr, const std::vector<const double*>& sourcePtrs,
                                    const std::array<int, 4>& targetSize,
                                    const std::vector<std::array<int, 4>>& sourceSizes,
                                    const std::vector<double>& scaleInputToNetInputs);
}
